#include "hip/hip_runtime.h"

#include "parameters.h"

#include "geo/init_geo.h"

#include "lbm/assit_func.h"
#include "lbm/init_block.h"
#include "lbm/force_block.h"
#include "lbm/error_L2.h"

#include "multi/init_multi.h"
#include "multi/FH/assit_multi.h"
#include "multi/FH/evolution_block.h"

#include "output/output.h"

#include "evolution.h"

int main()
{
	geo_init();

	Lbui N_varible;
	D2Q9 Variables_h[number_area], Variables_d[number_area];
	/*To allocate lbm variables*/
	for (Lbi a=0; a<number_area; a++)
	{
		N_varible = Areas[a].nx*Areas[a].ny;
		Variables_h[a].solid = (Lbi *)malloc(N_varible*sizeof(Lbi));
		hipMalloc((void **)&Variables_d[a].solid, N_varible*sizeof(Lbi));
		Lbd* *Vh = &Variables_h[a].x;
		Lbd* *Vd = &Variables_d[a].x;
		for (Lbi b=0; b<1+3*DMS+2*Q; b++)
		{
			Vh[b] = (Lbd *)malloc(N_varible*sizeof(Lbd));
			hipMalloc((void **)&Vd[b], N_varible*sizeof(Lbd)); 
		}
	}

	/*To allocate multi-block variables*/
	/*Temperal interpolation*/
	Ff_Store Line_T3_C[number_line>0 ? number_line : 1];
	for (Lbi a=0; a<number_line; a++)
	{ 
		Line_T3_C[a].marker[0] = 2;
		Line_T3_C[a].marker[1] = 1;
		Line_T3_C[a].marker[2] = 0;

		Line_T3_C[a].N = Line_Inf_C[a].Nf;

		F_k3 *ft = &Line_T3_C[a].f_t1;
		for (Lbi i=0; i<3; i++) //3t
		{
			Lbd **ftk = &ft[i].F[0];
			for (Lbi j=0; j<Q; j++) //Q
				hipMalloc((void **)&ftk[j], Line_T3_C[a].N*sizeof(Lbd));
		}
	}

	hipsparseHandle_t handle;
	hipsparseCreate(&handle);

	/*Spacial interpolation*/
	FM_spline3 Line_Spline3_C[number_line>0 ? number_line : 1];
	for (Lbi a=0; a<number_line; a++)
	{
		Line_Spline3_C[a].N = Line_Inf_C[a].Nc;

		hipMalloc((void **)&Line_Spline3_C[a].l, Line_Spline3_C[a].N*sizeof(Lbd));
		hipMalloc((void **)&Line_Spline3_C[a].m, Line_Spline3_C[a].N*sizeof(Lbd));
		hipMalloc((void **)&Line_Spline3_C[a].u, Line_Spline3_C[a].N*sizeof(Lbd));
		hipMalloc((void **)&Line_Spline3_C[a].Varibs_M, 12*Line_Spline3_C[a].N*sizeof(Lbd));

		Lbd* *fM = &Line_Spline3_C[a].rho;
		for (Lbi i=0; i<12; i++)
			hipMalloc((void **)&fM[i], Line_Spline3_C[a].N*sizeof(Lbd));
	}

	/*Get the block size of GPU kernels of LBM evolution*/
	dim3 block(_BLOCKX, _BLOCKY, 1);
	dim3 *grid = (dim3 *)malloc(number_area*sizeof(dim3));
	for (Lbi i=0; i<number_area; i++)
	{
		dim3 grid_tmp(int (ceil(Areas[i].nx/double (_BLOCKX))), int (ceil(Areas[i].ny/double (_BLOCKY))), 1);
		grid[i] = grid_tmp;
	} 

	/*To store the fluid velocity at the previous time step*/
	Lbd *upsx[number_area], *upsy[number_area];

	Lbd *h_uvp2_b[number_area];
	Lbd *d_uvp2_b[number_area];
	Lbd *h_uv2_b[number_area];
	Lbd *d_uv2_b[number_area];

	for (Lbi a=0; a<number_area; a++)
	{
		N_varible = Areas[a].nx*Areas[a].ny;
		hipMalloc((void **)&upsx[a], N_varible*sizeof(Lbd));
		hipMalloc((void **)&upsy[a], N_varible*sizeof(Lbd));

		Lbi Bx = int (ceil(Areas[a].nx/double (_BLOCKX)));
		Lbi By = int (ceil(Areas[a].ny/double (_BLOCKY)));
		Lbui size_B = Bx*By*sizeof(Lbd);

		h_uvp2_b[a] = (Lbd *)malloc(size_B);
		d_uvp2_b[a] = NULL;
		hipMalloc((void **)&d_uvp2_b[a], size_B);

		h_uv2_b[a] = (Lbd *)malloc(size_B);
		d_uv2_b[a] = NULL;
		hipMalloc((void **)&d_uv2_b[a], size_B);
	}

	/*To initialize the computational domain*/
	for (Lbi a=0; a<number_area; a++)
		init_General<<<grid[a], block>>>(Areas[a], Variables_d[a], upsx[a], upsy[a]);

	/*Output lbm variables*/
	output(0, Areas, Variables_h, Variables_d);
	
	/************************************************************************/

	/*To allocate variables for calculating the darg force with the momentum-exchange (ME) method*/
	Lbd Force[2];
	//Return the total number of lattices taking part in the ME
	Lbi N_drag = sum_N_drag(block, grid[number_area-1], Areas[number_area-1], Variables_d[number_area-1]);
	Int3 *d_Point_drag;
	hipMalloc((void **)&d_Point_drag, N_drag*sizeof(Int3));
	//Store the location (i, j ~ LX, LY) and the discrete velocity (k~Q) to ease the following drag force calculation
	fdrag_store(Areas[number_area-1], Variables_h[number_area-1], d_Point_drag, Variables_d[number_area-1], N_drag);
	
	/************************************************************************/

	/*To initialize the variables of multi-block*/
	for (Lbi a=0; a<number_line; a++)
	{
		init_Fstore<<< int (ceil(Line_T3_C[a].N/double (_BLOCK_1D))), _BLOCK_1D>>>(Areas[Line_Inf_C[a].areaf_n], Variables_d[Line_Inf_C[a].areaf_n], Line_Inf_C[a], Line_T3_C[a]);
		init_tri_A<<< int (ceil(Line_Spline3_C[a].N/double (_BLOCK_1D))), _BLOCK_1D>>>(Areas[Line_Inf_C[a].areac_n], Variables_d[Line_Inf_C[a].areac_n], Line_Inf_C[a], Line_Spline3_C[a]);
	}

	clock_t start, stop;
	Lbd duration;
	for (Lbi k=0; k<_LOOPS; k++)
	{
		duration = 0.0;
		start = clock();
		for (Lbi n=0; n<_T; n++)
		{
			/************************************************************************/
			evolution(k*_T+n, ASP, LS3CFFCP, LTCFFCP, LCCP, 0, block, grid, Areas, Variables_h, 
				Variables_d, Line_Inf_C, Line_Inf_F, Line_Inf_CC, Line_T3_C, Line_Spline3_C, handle);
			/************************************************************************/

			error_L2(k*_T+n, number_area, block, grid, Areas, Variables_d, upsx, upsy, h_uvp2_b, d_uvp2_b, h_uv2_b, d_uv2_b);

			getDrag_block(k*_T+n, d_Point_drag, Areas[number_area-1], Variables_d[number_area-1], Force, N_drag);
		}

		stop = clock();
		duration = (Lbd)(stop-start)/CLOCKS_PER_SEC;
		printf("%f seconds\n", duration);

		FILE *fp_t;
		fp_t=fopen("time_t.dat","a");
		fprintf(fp_t,"%d %g\n", (k+1)*_T, duration);
		fclose(fp_t);

		printf("%d\n", (k+1)*_T);
		output((k+1)*_T, Areas, Variables_h, Variables_d);
	}

	for (Lbi a=0; a<number_line; a++)
	{
		hipFree(Line_Spline3_C[a].l);
		hipFree(Line_Spline3_C[a].m);
		hipFree(Line_Spline3_C[a].u);
		hipFree(Line_Spline3_C[a].Varibs_M);
		Lbd* *fM = &Line_Spline3_C[a].rho;
		for (Lbi i=0; i<12; i++)
			hipFree(fM[i]);
	}
	hipsparseDestroy(handle);

	for (Lbi a=0; a<number_line; a++)
	{ 
		F_k3 *ft = &Line_T3_C[a].f_t1;
		for (Lbi i=0; i<3; i++) //3t
		{
			Lbd **ftk = &ft[i].F[0];
			for (Lbi j=0; j<Q; j++) //Q
				hipFree(ftk[j]);
		}
	}

	for (Lbi a=0; a<number_area; a++)
	{
		hipFree(upsx[a]);
		hipFree(upsy[a]);

		free(h_uvp2_b[a]);
		hipFree(d_uvp2_b[a]);

		free(h_uv2_b[a]);
		hipFree(d_uv2_b[a]);
	}
	/************************************************************************/
	
	hipFree(d_Point_drag);

	/************************************************************************/

	for (Lbi a=0; a<number_area; a++)
	{
		free(Variables_h[a].solid);
		hipFree(Variables_d[a].solid);
		Lbd* *Vh = &Variables_h[a].x;
		Lbd* *Vd = &Variables_d[a].x;
		for (Lbi b=0; b<1+3*DMS+2*Q; b++)
		{
			free(Vh[b]);
			Vh[b] = NULL;
			hipFree(Vd[b]); 
			Vd[b] = NULL;
		}
	}
	free(grid);
	hipDeviceReset();
	return 0; 
}